#include "hip/hip_runtime.h"
#include "render_gpu.hh"
#include "utils_gpu.cuh"

#define K 16

__global__ void nearest_neighbour(unsigned char *histos, unsigned width, unsigned height, size_t histos_pitch,
                                  float *centroids, size_t centroids_pitch, int *labels) {
    __shared__ unsigned min_index;
    __shared__ double min_dist;
    __shared__ double dist[K][HISTO_SIZE];

    if (threadIdx.x >= width || threadIdx.y >= height)
        return;

    if (threadIdx.x == 0) {
        min_dist = INFINITY;
    }
    __syncthreads();

    unsigned char *histo = histos + blockIdx.x * histos_pitch;
    double val = (double) histo[threadIdx.x];


    for (int i = 0; i < K; ++i) {
        float *centroid = centroids + i * centroids_pitch / sizeof(float);
        double diff = val - centroid[threadIdx.x];
        diff *= diff;
        dist[i][threadIdx.x] = diff;
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        for (int i = 0; i < K; ++i) {
            double dist_tmp = 0;
            for (int j = 0; j < blockDim.x; j++)
            {
                dist_tmp += dist[i][j];
            }
            if (min_dist > dist_tmp) {
                min_index = i;
                min_dist = dist_tmp;
            }
        }
        
        labels[blockIdx.x] = min_index;
    }
}

__global__ void colorize(unsigned char* image, unsigned width, unsigned height, size_t pitch,
                         const int *labels, const unsigned char *colors) {
    __shared__ int label;

    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        unsigned block_idx = (blockIdx.x + blockIdx.y * (width / blockDim.x));
        label = labels[block_idx];
    }

    __syncthreads();
    image[x * 3 + y * pitch] = colors[label * 3];
    image[x * 3 + y * pitch + 1] = colors[label * 3 + 1];
    image[x * 3 + y * pitch + 2] = colors[label * 3 + 2];
    __syncthreads();
}

unsigned char *render_gpu(unsigned width, unsigned height, unsigned char *histos,
                          size_t histos_pitch, float *centers, unsigned char *rand_colors) {
    hipError_t error;

    size_t block_size =(width / TILE_SIZE) * (height / TILE_SIZE);

    unsigned char *color_tab;
    error = hipMalloc(&color_tab, K * sizeof(unsigned char) * 3);
    if (error)
        abortError("Error on hipMalloc");

    error = hipMemcpy(color_tab, rand_colors, K * sizeof(unsigned char) * 3, hipMemcpyHostToDevice);
    if (error)
        abortError("Error on hipMemcpy2D");

    size_t pitch_centroids;
    float *centroids;

    error = hipMallocPitch(&centroids, &pitch_centroids, HISTO_SIZE * sizeof(float), K);
    if (error)
        abortError("Error on hipMallocPitch");

    error = hipMemcpy2D(centroids, pitch_centroids, centers, HISTO_SIZE * sizeof(float), HISTO_SIZE * sizeof(float),
                         K, hipMemcpyHostToDevice);
    if (error)
        abortError("Error on hipMemcpy2D");

    int *labels;
    error = hipMalloc(&labels, block_size * sizeof(int));
    if (error)
        abortError("Error on hipMalloc");

    nearest_neighbour<<<block_size, HISTO_SIZE>>>(histos, HISTO_SIZE, block_size, histos_pitch, centroids, pitch_centroids, labels);
    hipDeviceSynchronize();

    if(hipPeekAtLastError())
        abortError("Error on nearest neighbour kernel");

    unsigned char *image;
    size_t pitch_image;

    hipMallocPitch(&image, &pitch_image, width * sizeof(unsigned char) * 3, height);
    if (error)
        abortError("Error on mallocPitch");

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks(width / threads.x, height / threads.y);

    colorize<<<blocks, threads>>>(image, width, height, pitch_image, labels, color_tab);
    hipDeviceSynchronize();

    if(hipPeekAtLastError())
        abortError("Error on colorize kernel");

    error = hipFree(histos);
    if (error)
        abortError("Error on free");

    error = hipFree(centroids);
    if (error)
        abortError("Error on free");


    error = hipFree(color_tab);
    if (error)
        abortError("Error on free");

    error = hipFree(labels);
    if (error)
        abortError("Error on free");

    unsigned char *imageHost = (unsigned char *) malloc(width * height * sizeof(unsigned char) * 3);

    error = hipMemcpy2D(imageHost, width * sizeof(unsigned char) * 3, image, pitch_image,
                         width * sizeof(unsigned char) * 3, height, hipMemcpyDeviceToHost);
    if (error)
        abortError("Error on hipMemcpy2D");

    error = hipFree(image);
    if (error)
        abortError("Error on free");

    return imageHost;
}
